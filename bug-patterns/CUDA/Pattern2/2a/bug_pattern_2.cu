#include "hip/hip_runtime.h"
void __global__ kernel0(...,int device_N, int64_t* dNpairs,...)
{

  int64_t i =blockIdx.x*blockDim.x+threadIdx.x;

  if((i< N))
  {
    for(j=(i+1);j<N;j++) 
    {
      // other code
      if (flag) 
      {
        //other computation            
        dNpairs[mbin+1]++;
      } 
      //other calculation
    }
  }
}

/*second kernel*/
void __global__ kernel1(...)
{

  int64_t i =  blockIdx.x * blockDim.x + threadIdx.x;

  if(i==0)
  {
    // do element-wise      
    //reduction of dNpairs
  }
}

int main(){
 //other code
 kernel0<<<dimGrid,dimBlock>>>(..);
 kernel1<<<dimGrid,dimBlock>>>(..);
 //other code
}
